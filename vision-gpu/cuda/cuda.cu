#include "hip/hip_runtime.h"
// TODO can we replace some DeviceBox<T> with __device__ variables here?

#ifndef SMH_CONSTS
#include "../../vision-common/src/consts/consts.cu"
#endif

#include <hip/hip_math_constants.h>

#if __CUDA_ARCH__ < 600
#define atomicMax_block atomicMax
#define atomicMin_block atomicMin
#define atomicExch_block atomicExch
#define atomicAdd_block atomicAdd
#define atomicCAS_block atomicCAS
#endif

#define clamp(x, a, b) max((a), min((b), (x)))

#define XY(x, y) const uint32_t x, const uint32_t y
#define WH(w, h) const uint32_t w, const uint32_t h
#define XYWH(x, y, w, h) XY(x, y), WH(w, h)

#define LUMA_R 0.2126f
#define LUMA_G 0.7152f
#define LUMA_B 0.0722f
#define RGB8_TO_LUMA8(r, g, b) (uint8_t)(float(r) * LUMA_R + float(g) * LUMA_G + float(b) * LUMA_B)

#define PACK_1616_INTO_32(x, y) ((uint32_t)((uint32_t)(y) << 16) | (uint32_t)(x))
#define UNPACK_X_1616_FROM_32(xy) (uint16_t)((xy)&0xFFFF)
#define UNPACK_Y_1616_FROM_32(xy) (uint16_t)(((xy) >> 16) & 0xFFFF)

#define PACK_3232_INTO_64(x, y) ((uint64_t)((uint64_t)(y) << 32) | (uint64_t)(x))
#define UNPACK_X_3232_FROM_64(xy) (uint32_t)((xy)&0xFFFFFFFF)
#define UNPACK_Y_3232_FROM_64(xy) (uint32_t)(((xy) >> 32) & 0xFFFFFFFF)

#define IS_FIRST_THREAD threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0

#define INT_DIV_CEIL(self, rhs) (self + rhs - 1) / rhs

#ifdef ZERO_DEBUG
#define IS_NOT_ZERO(x) (x == 255)
#else
#define IS_NOT_ZERO(x) (x != 0)
#endif

__device__ __forceinline__ float atomicMax_block(float *const addr, const float value)
{
	float old;

	old = (value >= 0) ? __int_as_float(atomicMax_block((int *)addr, __float_as_int(value))) : __uint_as_float(atomicMin_block((unsigned int *)addr, __float_as_uint(value)));

	return old;
}

__device__ __forceinline__ float atomicCAS_block(float *const addr, const float compare, const float val)
{
	return __int_as_float(atomicCAS_block((int *const)addr, __float_as_int(compare), __float_as_int(val)));
}

__device__ uint8_t sat_subu8b(const uint8_t x, const uint8_t y)
{
	uint8_t res = x - y;
	res &= -(res <= x);

	return res;
}

__device__ uint8_t sat_addu8b(const uint8_t x, const uint8_t y)
{
	uint8_t res = x + y;
	res |= -(res < x);

	return res;
}

__device__ uint32_t sat_addu32b(const uint32_t x, const uint32_t y)
{
	uint32_t res = x + y;
	res |= -(res < x);

	return res;
}

__device__ uint16_t sat_subu16b(const uint16_t x, const uint16_t y)
{
	uint16_t res = x - y;
	res &= -(res <= x);

	return res;
}

extern "C" class HSV
{
public:
	// [0..360]
	uint16_t h;

	// [0..100]
	uint8_t s;

	// [0..100]
	uint8_t v;
};

extern "C" __align__(1) class RGB
{
public:
	uint8_t r, g, b;

	__device__ RGB() : r(0), g(0), b(0) {}
	__device__ RGB(const uint8_t r, const uint8_t g, const uint8_t b) : r(r), g(g), b(b) {}

	__device__ uint8_t operator[](uint8_t i) const
	{
#ifndef NDEBUG
		if (i > 2) [[unlikely]]
			return 0;
#endif
		return ((uint8_t *)this)[i];
	}

	__device__ uint8_t luma8() const
	{
		return RGB8_TO_LUMA8(r, g, b);
	}

	__device__ uint8_t mean() const
	{
		return (uint8_t)(((float)r + (float)g + (float)b) / 3.f);
	}

	static __device__ RGB from_hsv(HSV hsv)
	{
		float r, g, b;
		float h, s, v;

		h = (float)hsv.h;
		s = (float)hsv.s / 100.f;
		v = (float)hsv.v / 100.f;

		float f = h / 60.0f;
		float hi = floorf(f);
		f = f - hi;
		float p = v * (1.f - s);
		float q = v * (1.f - s * f);
		float t = v * (1.f - s * (1.f - f));

		if (hi == 0.0f || hi == 6.0f)
		{
			r = v;
			g = t;
			b = p;
		}
		else if (hi == 1.0f)
		{
			r = q;
			g = v;
			b = p;
		}
		else if (hi == 2.0f)
		{
			r = p;
			g = v;
			b = t;
		}
		else if (hi == 3.0f)
		{
			r = p;
			g = q;
			b = v;
		}
		else if (hi == 4.0f)
		{
			r = t;
			g = p;
			b = v;
		}
		else
		{
			r = v;
			g = p;
			b = q;
		}

		uint8_t red = (uint8_t)__float2uint_rn(255.0f * r);
		uint8_t green = (uint8_t)__float2uint_rn(255.0f * g);
		uint8_t blue = (uint8_t)__float2uint_rn(255.0f * b);
		return RGB{red, green, blue};
	}

	__device__ HSV to_hsv() const
	{
		float r, g, b;
		float h, s, v;

		r = this->r / 255.0f;
		g = this->g / 255.0f;
		b = this->b / 255.0f;

		float max = fmax(r, fmax(g, b));
		float min = fmin(r, fmin(g, b));
		float diff = max - min;

		v = max;

		if (v == 0.0f)
		{ // black
			h = s = 0.0f;
		}
		else
		{
			s = diff / v;
			if (diff < 0.001f)
			{ // grey
				h = 0.0f;
			}
			else
			{ // color
				if (max == r)
				{
					h = 60.0f * (g - b) / diff;
					if (h < 0.0f)
					{
						h += 360.0f;
					}
				}
				else if (max == g)
				{
					h = 60.0f * (2 + (b - r) / diff);
				}
				else
				{
					h = 60.0f * (4 + (r - g) / diff);
				}
			}
		}

		return HSV{(uint16_t)h, (uint8_t)(s * 100.f), (uint8_t)(v * 100.f)};
	}
};

extern "C" __align__(1) class BGRA
{
public:
	uint8_t b, g, r, a;

	__device__ __forceinline__ RGB to_rgb() const
	{
		return RGB(r, g, b);
	}
};

extern "C" struct RGBA
{
	uint8_t r, g, b, a;
};

extern "C" struct Point
{
	float x, y;
};

extern "C" struct Line
{
	Point p0, p1;
};

namespace markers
{
	extern "C" struct TemplateMatch
	{
		uint32_t xy;
		uint16_t sad;
	};
};

__device__ __forceinline__ bool is_map_marker_color(RGB rgb)
{
	constexpr uint16_t MAP_MARKER_COLORS[3][3] = {
		*ALPHA_MARKER_COLOR_HSV,
		*BRAVO_MARKER_COLOR_HSV,
		*CHARLIE_MARKER_COLOR_HSV};

	HSV hsv = rgb.to_hsv();

	bool alpha_hue_ok = abs((int16_t)hsv.h - (int16_t)ALPHA_MARKER_COLOR_HSV[0]) <= FIND_MARKER_HSV_HUE_TOLERANCE;
	bool alpha_sat_ok = abs((int16_t)hsv.s - (int16_t)ALPHA_MARKER_COLOR_HSV[1]) <= FIND_MARKER_HSV_SAT_TOLERANCE;
	bool alpha_vib_ok = abs((int16_t)hsv.v - (int16_t)ALPHA_MARKER_COLOR_HSV[2]) <= FIND_MARKER_HSV_VIB_TOLERANCE;

	bool bravo_hue_ok = abs((int16_t)hsv.h - (int16_t)BRAVO_MARKER_COLOR_HSV[0]) <= FIND_MARKER_HSV_HUE_TOLERANCE;
	bool bravo_sat_ok = abs((int16_t)hsv.s - (int16_t)BRAVO_MARKER_COLOR_HSV[1]) <= FIND_MARKER_HSV_SAT_TOLERANCE;
	bool bravo_vib_ok = abs((int16_t)hsv.v - (int16_t)BRAVO_MARKER_COLOR_HSV[2]) <= FIND_MARKER_HSV_VIB_TOLERANCE;

	bool charlie_hue_ok = abs((int16_t)hsv.h - (int16_t)CHARLIE_MARKER_COLOR_HSV[0]) <= FIND_MARKER_HSV_HUE_TOLERANCE;
	bool charlie_sat_ok = abs((int16_t)hsv.s - (int16_t)CHARLIE_MARKER_COLOR_HSV[1]) <= FIND_MARKER_HSV_SAT_TOLERANCE;
	bool charlie_vib_ok = abs((int16_t)hsv.v - (int16_t)CHARLIE_MARKER_COLOR_HSV[2]) <= FIND_MARKER_HSV_VIB_TOLERANCE;

	return (alpha_hue_ok &&
			alpha_sat_ok &&
			alpha_vib_ok) ||
		   (bravo_hue_ok &&
			bravo_sat_ok &&
			bravo_vib_ok) ||
		   (charlie_hue_ok &&
			charlie_sat_ok &&
			charlie_vib_ok);
}

// Counts the number of red pixels where the "CLOSE DEPLOYMENT BUTTON" is on the screen
extern "C" __global__ void count_close_deployment_button_red_pixels(
	const BGRA *const input,
	const uint32_t stride,
	XYWH(btn_x, btn_y, btn_w, btn_h),
	uint32_t *const red_pixels)
{
	__shared__ uint32_t block_red_pixels;

	if (threadIdx.x == 0 && threadIdx.y == 0) [[unlikely]]
		block_red_pixels = 0;

	__threadfence_block();

	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < btn_w && y < btn_h) [[likely]]
	{
		const unsigned int btn_roi_x = x + btn_x;
		const unsigned int btn_roi_y = y + btn_y;

		const RGB px = input[btn_roi_y * stride + btn_roi_x].to_rgb();

		bool passed = true;
		for (uint8_t i = 0; i < 3; i++)
		{
			if ((uint16_t)abs(CLOSE_DEPLOYMENT_BUTTON_COLOR[i] - (int16_t)px[i]) > CLOSE_DEPLOYMENT_BUTTON_TOLERANCE)
			{
				passed = false;
			}
		}

		if (passed)
			atomicAdd_block(&block_red_pixels, 1);
	}

	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0) [[unlikely]]
		atomicAdd(red_pixels, block_red_pixels);
}

extern "C" __global__ void crop_to_map_brq(
	const BGRA *const input,
	const uint32_t stride,
	XYWH(roi_x, roi_y, roi_w, roi_h),
	RGB *const output)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= roi_w || y >= roi_h || x >= stride) [[unlikely]]
		return;

	output[y * roi_w + x] = input[(y + roi_y) * stride + (x + roi_x)].to_rgb();
}

extern "C" __global__ void crop_to_map_grayscale(
	const BGRA *const input,
	const uint32_t stride,
	XYWH(roi_x, roi_y, roi_w, roi_h),
	RGB *const output,
	RGBA *const gray_output)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= roi_w || y >= roi_h || x >= stride) [[unlikely]]
		return;

	const RGB pixel = input[(y + roi_y) * stride + (x + roi_x)].to_rgb();
	output[y * roi_w + x] = pixel;

	const uint8_t luma8 = pixel.luma8();
	gray_output[y * roi_w + x] = RGBA{luma8, luma8, luma8, 255};
}

extern "C" __global__ void crop_to_map(
	const BGRA *const input,
	const uint32_t stride,
	XYWH(roi_x, roi_y, roi_w, roi_h),
	RGB *const output,
	RGBA *const ui_output)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= roi_w || y >= roi_h || x >= stride) [[unlikely]]
		return;

	const RGB pixel = input[(y + roi_y) * stride + (x + roi_x)].to_rgb();
	output[y * roi_w + x] = pixel;

	ui_output[y * roi_w + x] = RGBA{pixel.r, pixel.g, pixel.b, 255};
}

// Isolate whiteish text
// We don't use binary thresholding here because the OCR reads
// antialiasing better than we can threshold it
extern "C" __global__ void ocr_preprocess(
	const RGB *const input,
	WH(w, h),
	uint8_t *const out)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= w || y >= h) [[unlikely]]
		return;

	const RGB pixel = input[y * w + x];

	const auto ocr_monochromaticy = [](const RGB pixel)
	{
		uint16_t diff = 0;
		for (uint8_t i = 0; i < 3; i++)
		{
			for (uint8_t j = 0; j < 3; j++)
			{
				diff += abs((int16_t)pixel[i] - (int16_t)pixel[j]);
			}
		}
		return diff;
	};

	const uint16_t diff = ocr_monochromaticy(pixel);
	if (diff <= OCR_PREPROCESS_MONOCHROMATICY_THRESHOLD)
	{
		for (uint8_t i = 0; i < 3; i++)
		{
			if (pixel[i] < OCR_PREPROCESS_BRIGHTNESS_THRESHOLD)
			{
				goto edge;
			}
		}
		goto keep;
	}

edge:
	if (diff <= OCR_PREPROCESS_SIMILARITY_EDGE_THRESHOLD)
	{
		for (uint8_t i = 0; i < 3; i++)
		{
			if (pixel[i] < OCR_PREPROCESS_BRIGHTNESS_EDGE_THRESHOLD)
			{
				goto dont_keep;
			}
		}

		for (int32_t xx = x - OCR_PREPROCESS_DILATE_RADIUS; xx <= x + OCR_PREPROCESS_DILATE_RADIUS; xx++)
		{
			for (int32_t yy = y - OCR_PREPROCESS_DILATE_RADIUS; yy <= y + OCR_PREPROCESS_DILATE_RADIUS; yy++)
			{
				if (xx < 0 || xx >= w || yy < 0 || yy >= h)
					continue;

				const RGB pixel = input[yy * w + xx];

				for (uint8_t i = 0; i < 3; i++)
				{
					if (pixel[i] < OCR_PREPROCESS_BRIGHTNESS_THRESHOLD)
					{
						goto next_neighbour;
					}
				}

				if (ocr_monochromaticy(pixel) <= OCR_PREPROCESS_MONOCHROMATICY_THRESHOLD)
				{
					goto keep;
				}

			next_neighbour:
			}
		}
	}

dont_keep:
	out[y * w + x] = 255;
	return;

keep:
	out[y * w + x] = 255 - pixel.luma8();
}

extern "C" __global__ void find_scales_preprocess(
	const RGB *const input,
	WH(w, h),
	const uint32_t scales_start_y,
	uint8_t *const output)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = (threadIdx.y + blockIdx.y * blockDim.y) + scales_start_y;

	if (x >= w || y >= h) [[unlikely]]
		return;

	// Only need black & white pixels
	if (IS_NOT_ZERO(input[y * w + x].luma8()))
	{
		output[y * w + x] = 255;
	}
	else
	{
		output[y * w + x] = 0;
	}
}

extern "C" __global__ void isolate_map_markers(
	RGB *const input,
	WH(w, h),

	markers::TemplateMatch *const marked_map_marker_pixels,
	uint32_t *const marked_map_marker_pixels_count,
	const uint32_t marker_size)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= w || y >= h) [[unlikely]]
		return;

	if (!is_map_marker_color(input[y * w + x]))
	{
		input[y * w + x] = RGB(0, 0, 0);
	}
	else if (x < w - marker_size && y < h - marker_size)
	{
		marked_map_marker_pixels[atomicAdd(marked_map_marker_pixels_count, 1)] = markers::TemplateMatch{
			y * w + x,
			0};
	}
}

extern "C" __global__ void filter_map_marker_icons(
	RGB *const input,
	const uint32_t stride,

	markers::TemplateMatch *const marked_map_marker_pixels,

	const RGBA **const markers,
	const uint32_t marker_size,

	const uint32_t markers_n,
	const uint32_t marked_map_marker_pixels_count)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= markers_n || y >= marked_map_marker_pixels_count) [[unlikely]]
		return;

	const RGBA *const marker = markers[x];

	markers::TemplateMatch &template_match = marked_map_marker_pixels[y];

	const uint32_t xx = template_match.xy % stride;
	const uint32_t yy = template_match.xy / stride;

	for (uint32_t marker_x = 0; marker_x < marker_size; marker_x++)
	{
		for (uint32_t marker_y = 0; marker_y < marker_size; marker_y++)
		{
			RGBA marker_pixel = marker[marker_y * marker_size + marker_x];
			RGB pixel = input[(yy + marker_y) * stride + (xx + marker_x)];

			uint16_t ad = (uint16_t)abs((int16_t)pixel.r - (int16_t)marker_pixel.r) + (uint16_t)abs((int16_t)pixel.g - (int16_t)marker_pixel.g) + (uint16_t)abs((int16_t)pixel.b - (int16_t)marker_pixel.b);
			ad = (float)ad * ((float)marker_pixel.a / 255.0); // alpha blending
			template_match.sad += ad;
		}
	}
}

extern "C" __global__ void filter_map_marker_icons_clear(
	RGB *const input,
	WH(w, h),

	const uint32_t min_sad_xy,
	const uint32_t map_marker_size)
{
	const uint32_t roi_x = min_sad_xy % w;
	const uint32_t roi_y = min_sad_xy / w;

	const unsigned int x = (threadIdx.x + blockIdx.x * blockDim.x) + roi_x;
	const unsigned int y = (threadIdx.y + blockIdx.y * blockDim.y) + roi_y;

	if (x >= w || y >= h) [[unlikely]]
		return;

	// Erase the marker icon from the map!
	input[y * w + x] = RGB(0, 0, 0);

	// Trick the line segment detection algorithm into continuing the line by placing a 4x4 square where the marker icon was pointing
	// It should hopefully fill the gap and continue the line
	const uint32_t sq_x = roi_x + (map_marker_size / 2);
	const uint32_t sq_y = roi_y + roundf((float)map_marker_size * MAP_MARKER_POI_LOCATION);
	if (x >= sq_x - 2 && x <= sq_x + 2 && y >= sq_y - 2 && y <= sq_y + 2)
	{
		input[y * w + x] = RGB(0, 255, 0);
	}
}

extern "C" __global__ void mask_marker_lines(
	const RGB *const input,
	WH(w, h),
	uint8_t *const output)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= w || y >= h) [[unlikely]]
		return;

	if (is_map_marker_color(input[y * w + x]))
	{
		output[y * w + x] = 255;
	}
	else
	{
		output[y * w + x] = 0;
	}
}

extern "C" __global__ void find_longest_line(
	const uint8_t *const input,
	WH(w, h),

	const Point pt,
	const float max_gap,

	Line *const longest_lines)
{
	__shared__ float longest_line_length;
	if (threadIdx.x == 0)
	{
		longest_line_length = 0.0f;
	}

	const float theta = ((float)(threadIdx.x + blockIdx.x * blockDim.x) / 10.0) * HIP_PI_F / 180.0;

	float x = pt.x;
	float y = pt.y;

	const float x_start = x;
	const float y_start = y;
	float x_end = x;
	float y_end = y;

	float gap = 0.0;
	float gap_x = 0.0;
	float gap_y = 0.0;

	const float dx = cosf(theta);
	const float dy = sinf(theta);
	float x_offset = 0.0;
	float y_offset = 0.0;

	while (x >= 0.0 && y >= 0.0 && x < w && y < h)
		[[likely]]
		{
			if (input[(uint32_t)y * w + (uint32_t)x] == 255)
			{
				// there's no gap, reset state
				gap = 0.0;
				gap_x = 0.0;
				gap_y = 0.0;
			}
			else if (gap >= max_gap)
			{
				// gap didn't close, abort
				// restore saved state
				x = gap_x;
				y = gap_y;
				break;
			}
			else if (gap == 0.0)
			{
				// save the state of (x, y) so we can restore it later if the gap isn't closed
				gap = 1.0;
				gap_x = x;
				gap_y = y;
			}
			else
			{
				// keep going in case there is a gap that closes
				gap += 1.0;
			}

			x_offset += dx;
			y_offset += dy;
			x = x_offset + x_start;
			y = y_offset + y_start;
		}

	if ((uint32_t)x < w && (uint32_t)y < h && input[(uint32_t)y * w + (uint32_t)x] == 0)
	{
		x_end = x - dx;
		y_end = y - dy;
	}

	const Line line = Line{
		Point{x_start, y_start},
		Point{x_end, y_end}};

	const float length = ((line.p0.x - line.p1.x) * (line.p0.x - line.p1.x)) + ((line.p0.y - line.p1.y) * (line.p0.y - line.p1.y));

	atomicMax_block(&longest_line_length, length);

	__syncthreads();

	// only one thread with the longest line can write to the output
	const bool claim = atomicCAS_block(&longest_line_length, length, -1.0) == length;
	if (claim) [[unlikely]]
	{
		longest_lines[blockIdx.x] = line;
	}
}